#include "hip/hip_runtime.h"
#include "main.h"

#define CTA_SIZE 4

__device__ uint32_t getBin(uint32_t val, uint32_t bit, uint32_t nBins) {
    return (val >> bit) & (nBins - 1);
}

__global__ void computeHistKernel(uint32_t * in, int n, uint32_t * hist, int nBins, int bit, int gridSize) {
    extern __shared__ int s_hist[];
    for (int idx = threadIdx.x; idx < nBins; idx += blockDim.x)
        s_hist[idx] = 0;
    __syncthreads();

    // Each block computes its local hist using atomic on SMEM
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        atomicAdd(&s_hist[getBin(in[i], bit, nBins)], 1);
    __syncthreads();

    // Each block adds its local hist to global hist using atomic on GMEM
    for (int digit = threadIdx.x; digit < nBins; digit += blockDim.x)
        hist[blockIdx.x + digit * gridSize] = s_hist[digit];
}

__global__ void scanBlkKernel(uint32_t * in, int n, uint32_t * out, uint32_t * blkSums) {
    extern __shared__ uint32_t s_in[];
    int id_in = blockDim.x * blockIdx.x + threadIdx.x;
    s_in[threadIdx.x] = id_in < n ? in[id_in] : 0;
    __syncthreads();

    int turn = 0;
    for (int stride = 1; stride < blockDim.x; stride <<= 1) {
        turn ^= 1;
        uint32_t cur = s_in[threadIdx.x + (turn ^ 1) * blockDim.x];
        if (threadIdx.x >= stride)
            cur += s_in[threadIdx.x - stride + (turn ^ 1) * blockDim.x]; 
        s_in[threadIdx.x + turn * blockDim.x] = cur;
        __syncthreads();
    }

    if (threadIdx.x == blockDim.x - 1) { // last thread
        blkSums[blockIdx.x] = s_in[threadIdx.x + turn * blockDim.x];
    }

    if (id_in < n) {
        out[id_in] = s_in[threadIdx.x + turn * blockDim.x];
    }
}

__global__ void sumPrefixBlkKernel(uint32_t * out, int n, uint32_t * blkSums) {
    int id_in = blockDim.x * blockIdx.x + threadIdx.x;
    if (id_in < n && blockIdx.x > 0) {
        out[id_in] += blkSums[blockIdx.x - 1];
    }
}

__global__ void reduceKernel(uint32_t * in, int n, uint32_t * out) {
    int id_in = blockDim.x * blockIdx.x + threadIdx.x;
    if (id_in < n)
        out[id_in] -= in[id_in];
}

void computeScanArray(uint32_t* d_in, uint32_t* d_out, int n, dim3 blkSize) {
    dim3 gridSize((n - 1) / blkSize.x + 1);

    uint32_t * d_blkSums;
    CHECK(hipMalloc(&d_blkSums, gridSize.x * sizeof(uint32_t)));
    uint32_t * d_sum_blkSums;
    CHECK(hipMalloc(&d_sum_blkSums, gridSize.x * sizeof(uint32_t)));

    scanBlkKernel<<<gridSize, blkSize, 2 * blkSize.x * sizeof(uint32_t)>>>
        (d_in, n, d_out, d_blkSums);
    if (gridSize.x != 1) {
        computeScanArray(d_blkSums, d_sum_blkSums, gridSize.x, blkSize);
    }
    sumPrefixBlkKernel<<<gridSize, blkSize>>>(d_out, n, d_sum_blkSums);

    CHECK(hipFree(d_sum_blkSums));
    CHECK(hipFree(d_blkSums));
}

__device__ uint32_t* countEqualBefore(uint32_t* src, uint32_t* buffer, int bit, int nBins) {
    uint32_t thisSrc = src[threadIdx.x], thisBin = getBin(thisSrc, bit, nBins);
    buffer[threadIdx.x] = 1;
    __syncthreads();
    int turn = 0;
    for (int stride = 1; stride < blockDim.x; stride <<= 1) {
        turn ^= 1;
        uint32_t cur = buffer[threadIdx.x + (turn ^ 1) * blockDim.x];
        if (threadIdx.x >= stride &&  thisBin == getBin(src[threadIdx.x - stride], bit, nBins))
            cur += buffer[threadIdx.x - stride + (turn ^ 1) * blockDim.x]; 
        buffer[threadIdx.x + turn * blockDim.x] = cur;
        __syncthreads();
    }
    return buffer + turn * blockDim.x;
}

__global__ void scatterKernel(uint32_t* src, int n, uint32_t* dst, uint32_t* histScan, int bit, int nBins, int gridSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ uint32_t s[];
    uint32_t * localSrc = s;
    uint32_t * localBuffer = localSrc + blockDim.x;
    
    localSrc[threadIdx.x] = i < n ? src[i] : UINT_MAX;

    uint32_t* count = countEqualBefore(localSrc, localBuffer, bit, nBins); 
    
    // scatter
    uint32_t pos =
        histScan[blockIdx.x + getBin(localSrc[threadIdx.x], bit, nBins) * gridSize]
        + count[threadIdx.x]
        - 1;
    
    if (pos < n) {
        dst[pos] = localSrc[threadIdx.x];
    }
}

__global__ void sortLocalKernel(uint32_t* src, int n, uint32_t* dst, int bit, int k) {
    extern __shared__ uint32_t s[];
    uint32_t * localSrc = s;
    uint32_t * localScan = localSrc + CTA_SIZE * blockDim.x;

    int id_in = CTA_SIZE * (blockDim.x * blockIdx.x + threadIdx.x);
    for (int i = 0; i < CTA_SIZE; ++i)
        localSrc[CTA_SIZE * threadIdx.x + i] = (id_in + i < n ? src[id_in + i] : UINT_MAX);

    for (int blockBit = bit; blockBit < bit + k; ++blockBit) {
        uint32_t temp[CTA_SIZE];
        uint32_t val = 0;
        for (int i = 0; i < CTA_SIZE; ++i) {
            temp[i] = localSrc[CTA_SIZE * threadIdx.x + i]; 
            val += (temp[i] >> blockBit & 1);
        }
        // compute scan
        localScan[threadIdx.x] = val;
        __syncthreads();
        int turn = 0;
        for (int stride = 1; stride < blockDim.x; stride <<= 1) {
            turn ^= 1;
            uint32_t cur = localScan[threadIdx.x + (turn ^ 1) * blockDim.x];
            if (threadIdx.x >= stride)
                cur += localScan[threadIdx.x - stride + (turn ^ 1) * blockDim.x]; 
            localScan[threadIdx.x + turn * blockDim.x] = cur;
            __syncthreads();
        }
        
        // scatter
        int n0 = CTA_SIZE * blockDim.x - localScan[blockDim.x - 1 + turn * blockDim.x];
        val = localScan[threadIdx.x + turn * blockDim.x];
        for (int i = CTA_SIZE - 1; i >= 0; --i) {
            if (temp[i] >> blockBit & 1)
                localSrc[n0 + val - 1] = temp[i];
            else
                localSrc[CTA_SIZE * threadIdx.x + i - val] = temp[i];
            val -= (temp[i] >> blockBit & 1);
        }
        __syncthreads();
    }

    for (int i = 0; i < CTA_SIZE; ++i)
        if (id_in + i < n)
            src[id_in + i] = localSrc[CTA_SIZE * threadIdx.x + i];
}

void sort(const uint32_t * in, int n, uint32_t * out, int k, int blkSize) {
    int nBins = 1 << k;
    uint32_t * d_src;
    uint32_t * d_dst;
    uint32_t * d_hist;
    uint32_t * d_histScan;
    CHECK(hipMalloc(&d_src, n * sizeof(uint32_t)));
    CHECK(hipMemcpy(d_src, in, n * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_dst, n * sizeof(uint32_t)));

    // Compute block and grid size for scan and scatter phase
    dim3 blockSize(blkSize);
    dim3 blockSizeCTA(blkSize / CTA_SIZE);
    dim3 gridSize((n - 1) / blockSize.x + 1);

    int histSize = nBins * gridSize.x;
    CHECK(hipMalloc(&d_hist, histSize * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_histScan, histSize * sizeof(uint32_t)));
    dim3 gridSizeScan((histSize - 1) / blockSize.x + 1);

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += k) {
        // compute hist
        computeHistKernel<<<gridSize, blockSize, nBins * sizeof(uint32_t)>>>
            (d_src, n, d_hist, nBins, bit, gridSize.x);
        
        // compute hist scan
        computeScanArray(d_hist, d_histScan, histSize, blockSize);
        reduceKernel<<<gridSizeScan, blockSize>>>
            (d_hist, histSize, d_histScan);
        
        // scatter
        sortLocalKernel<<<gridSize, blockSizeCTA, (CTA_SIZE + 2) * blockSizeCTA.x * sizeof(uint32_t)>>>
            (d_src, n, d_dst, bit, k);
        scatterKernel<<<gridSize, blockSize, 3 * blockSize.x * sizeof(uint32_t)>>>
            (d_src, n, d_dst, d_histScan, bit, nBins, gridSize.x);
        
        uint32_t * tmp = d_src; d_src = d_dst; d_dst = tmp;
    }

    CHECK(hipMemcpy(out, d_src, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
    
    CHECK(hipFree(d_src));
    CHECK(hipFree(d_dst));
    CHECK(hipFree(d_hist));
    CHECK(hipFree(d_histScan));
}